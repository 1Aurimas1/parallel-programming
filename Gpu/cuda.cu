#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <iomanip>
// Additional libraries used:
// https://github.com/nlohmann/json
#include "json.hpp"

#define INPUT_FILE "./input.json"
#define OUTPUT_FILE "./output.txt"

using namespace std;
using json = nlohmann::json;

const int MAX_CHAR_LENGTH = 200;
const int THREADS = 32;
const int BLOCKS = 2;

struct Car {
    char name[MAX_CHAR_LENGTH];
    int year;
    double mileage;
	int calculatedValue;

	void from_json(json data) {
		string tempName = data["name"];

		strncpy(name, tempName.c_str(), sizeof(name));
		name[sizeof(name) - 1] = 0;
		year = data["year"];
		mileage = data["mileage"];
	}
};


vector<Car> readFile() {
    vector<Car> cars;

    ifstream f(INPUT_FILE);
    json fileData = json::parse(f);

    json carsData = fileData["cars"];
    for (int i = 0; i < carsData.size(); i++) {
		Car tempCar;
		tempCar.from_json(carsData[i]);
        cars.push_back(tempCar);
    }

    return cars;
}

__device__ void gpuMemset(char* dest) {
    for (int i = 0; i < MAX_CHAR_LENGTH; ++i) {
        dest[i] = 0;
    }
}

__device__ void gpuStrcat(char* src, char* dest) {
    for (int i = 0; i < MAX_CHAR_LENGTH; ++i) {
        if (dest[i] == 0) {
            for (int j = 0; j < MAX_CHAR_LENGTH; ++j) {
                if (src[j] != 0) {
                    dest[i + j] = src[j];
                }
            }
            break;
        }
    }
}

__global__ void runOnGpu(Car* cars, int* n, int* blockSize, int* chunkSize, Car* results, int* resSize) {
	int startIdx = (blockIdx.x * *blockSize) + threadIdx.x * *chunkSize;
	int endIdx = ((blockIdx.x) * *blockSize) + (threadIdx.x + 1) * *chunkSize;

	if (endIdx + (*chunkSize * BLOCKS) >= *n) {
        endIdx = *n;
    }

    for (int i = startIdx; i < endIdx; i++) {
		Car resultCar;
		gpuMemset(resultCar.name);
		gpuStrcat(cars[i].name, resultCar.name);
		resultCar.year = cars[i].year;
		resultCar.mileage = cars[i].mileage;
		
		int year = resultCar.year;
		year *= 100;
		if (year >= resultCar.mileage) {
			resultCar.calculatedValue = year;
			int idx = atomicAdd(resSize, 1);
			results[idx] = resultCar;
		}
    } 
}

void writeResults(Car* results, int size) {
    const int NameWidth = 25;
    const int YearWidth = 5;
    const int MileageWidth = 10;
    const int ValWidth = 16;
    const char HorizontalSeparator = '-';
    const int HLength = NameWidth + YearWidth + MileageWidth + ValWidth + 11;
    const char* VerticalSeparator = " | ";

    ofstream output(OUTPUT_FILE);

    output << left << "Results" << endl;
    output << string(HLength, HorizontalSeparator) << endl;
    output << right << setw(NameWidth) << "Name" << VerticalSeparator
        << setw(YearWidth) << "Year" << VerticalSeparator
        << setw(MileageWidth) << "Mileage" << VerticalSeparator
        << setw(ValWidth) << "Calculated value" << VerticalSeparator << endl;
    output << string(HLength, HorizontalSeparator) << endl;
    for (int i = 0; i < size; i++)
    {
        Car car = results[i];
        output << right << setw(NameWidth) << car.name << VerticalSeparator << setw(YearWidth)
            << car.year << VerticalSeparator << fixed << setprecision(2) << setw(MileageWidth)
            << car.mileage << VerticalSeparator << setw(ValWidth) << car.calculatedValue
            << VerticalSeparator << endl;
    }
    output << string(HLength, HorizontalSeparator) << endl;

    output.close();
}


int main() {

	vector<Car> inputVec = readFile();

	Car* cars = &inputVec[0];
	int arrSize = inputVec.size();
	int resSize = 0;

	Car results[arrSize];
	int chunkSizePerBlock = arrSize / BLOCKS / THREADS;
	int blockSize = arrSize / BLOCKS;

	Car* deviceCars;
	Car* deviceResults;
	int* deviceN;
	int* deviceChunkSize;
	int* deviceBlockSize;
	int* deviceResultsN;
	
	hipMalloc((void**)&deviceCars, arrSize * sizeof(Car));
	hipMalloc((void**)&deviceResults, arrSize * sizeof(Car));
	hipMalloc((void**)&deviceN, sizeof(int));
	hipMalloc((void**)&deviceChunkSize, sizeof(int));
	hipMalloc((void**)&deviceBlockSize, sizeof(int));
	hipMalloc((void**)&deviceResultsN, sizeof(int));
	
	hipMemcpy(deviceCars, cars, arrSize * sizeof(Car), hipMemcpyHostToDevice);
	hipMemcpy(deviceN, &arrSize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceChunkSize, &chunkSizePerBlock, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceBlockSize, &blockSize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceResultsN, &resSize, sizeof(int), hipMemcpyHostToDevice);
	
	runOnGpu<<<BLOCKS, THREADS>>>(deviceCars, deviceN, deviceBlockSize, deviceChunkSize, deviceResults, deviceResultsN);
	hipDeviceSynchronize();
	
	hipMemcpy(&results, deviceResults, arrSize * sizeof(Car), hipMemcpyDeviceToHost);
	hipMemcpy(&resSize, deviceResultsN, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(deviceCars);
	hipFree(deviceN);
	hipFree(deviceChunkSize);
	hipFree(deviceResults);
	hipFree(deviceBlockSize);

    hipDeviceSynchronize();

	writeResults(results, resSize);
	
	return 0;
}